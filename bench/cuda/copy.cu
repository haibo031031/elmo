#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "util.h"

#define BLOCK_SIZE 512

//__global__ void copy(float * in, float * out, int size)
__global__ void copy(float * out)  
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	
	__syncthreads();
	int a_0 = tid * 0;
	int a_1 = tid * 1 + a_0;
	int a_2 = tid * 2 + a_1;
	int a_3 = tid * 3 + a_2;
	int a_4 = tid * 4 + a_3;
	int a_5 = tid * 5 + a_4;
	__syncthreads();
	
	int val = a_0 - a_1 + a_2 - a_3 + a_4 - a_5;
	out[tid] = val;
}

int main(int argc, char* argv[]) 
{
	bool verify = true;
	/* memory allocation*/	
	int w = 1024, h = 1024;
	int size = w * h;
	float * h_in = (float *)malloc(size * sizeof(float));
	float * h_out = (float *)malloc(size * sizeof(float));
	float * d_in, * d_out;
	hipMalloc(&d_in, size * sizeof(float));
	hipMalloc(&d_out, size*sizeof(float));
	
	/* transfer data from host(cpu) to device(gpu) */
	hipMemcpy(d_in, h_in, size * sizeof(float), hipMemcpyHostToDevice);

	/* kernel execution */
	int work_items = size;
	if(work_items % BLOCK_SIZE != 0)
	{
		work_items = size + (BLOCK_SIZE - size % BLOCK_SIZE);
	}
	//copy<<<work_items/BLOCK_SIZE, BLOCK_SIZE>>>(d_in, d_out, size);
	copy<<<work_items/BLOCK_SIZE, BLOCK_SIZE>>>(d_out);
	hipError_t e = hipGetLastError();
	if(e != hipSuccess)
	{
		printf("failure in kernel::%s\n", hipGetErrorString(e));
		exit(1);
	}	
	hipDeviceSynchronize();

	/* transfer data from gpu to cpu */
	hipMemcpy(h_out, d_out, size * sizeof(float), hipMemcpyDeviceToHost);

	if(verify)
	{
		verify_array<float>(h_in, h_out, size);
	}	
	
	hipFree(d_in);
	hipFree(d_out);
	if(h_in != 0) free(h_in);
	if(h_out != 0) free(h_out);
	
	return 0;
}

